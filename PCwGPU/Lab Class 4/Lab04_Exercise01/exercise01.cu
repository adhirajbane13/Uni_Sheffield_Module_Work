#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


//The number of character in the encrypted text
#define N 1024

void checkCUDAError(const char*);
void read_encrypted_file(int*);


/* Exercise 1.1 */
__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void affine_decrypt(int *d_input, int *d_output)
{
	/* Exercise 1.2 */
	int t_ind = threadIdx.x;
	if (t_ind < N) {
		int encrypted_char = d_input[t_ind];
		int decrypted_char = modulo(111 * (encrypted_char - 27), 128);
		d_output[t_ind] = decrypted_char;
	}
}

__global__ void affine_decrypt_multiblock(int *d_input, int *d_output)
{
	/* Exercise 1.8 */
		int t_ind = blockIdx.x * blockDim.x + threadIdx.x;
		if (t_ind < N) {
			int encrypted_char = d_input[t_ind];
			int decrypted_char = modulo(111 * (encrypted_char - 27), 128);
			d_output[t_ind] = decrypted_char;
		}
}


int main(int argc, char *argv[])
{
	int *h_input, *h_output;
	int *d_input, *d_output;
	unsigned int size;
	int i;

	size = N * sizeof(int);

	/* allocate the host memory */
	h_input = (int *)malloc(size);
	h_output = (int *)malloc(size);

	/* Exercise 1.3: allocate device memory */
	//cudaMalloc(???);
	//cudaMalloc(???);
	hipMalloc((void**)&d_input, size);
	hipMalloc((void**)&d_output, size);
	checkCUDAError("Memory allocation");

	/* read the encryted text */
	read_encrypted_file(h_input);

	/* Exercise 1.4: copy host input to device input */
	//cudaMemcpy(???);
	hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
	checkCUDAError("Input transfer to device");

	/* Exercise 1.5: Configure the grid of thread blocks and run the GPU kernel */
	//dim3 blocksPerGrid(???);
	//dim3 threadsPerBlock(???);
	//affine_decrypt(???);
	//dim3 blocksPerGrid(1);
	//dim3 threadsPerBlock(N);
	dim3 blocksPerGrid(8);
	dim3 threadsPerBlock(128);
	affine_decrypt_multiblock <<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);


	/* wait for all threads to complete */
	hipDeviceSynchronize();
	checkCUDAError("Kernel execution");

	/* Exercise 1.6: copy the gpu output back to the host */
	//cudaMemcpy(???);
	hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

	checkCUDAError("Result transfer to host");

	/* print out the result to screen */
	for (i = 0; i < N; i++) {
		printf("%c", (char)h_output[i]);
	}
	printf("\n");

	/* Exercise 1.7: free device memory */
	//cudaFree(???);
	//cudaFree(???);
	hipFree(d_input);
	hipFree(d_output);

	checkCUDAError("Free memory");

	/* free host buffers */
	free(h_input);
	free(h_output);

	return 0;
}


void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void read_encrypted_file(int* input)
{
	FILE *f = NULL;
	f = fopen("encrypted01.bin", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find encrypted01.bin file \n");
		exit(1);
	}
	//read encrypted data
	fread(input, sizeof(unsigned int), N, f);
	fclose(f);
}
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define N 65536
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int* a);



__global__ void vectorAdd(int* a, int* b, int* c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}



int main(void) {
	//int *a,*b,*c,*c_ref;
	//int a[N], b[N], c[N], c_ref[N]; // Static arrays with size N; host copies of a, b, c
	int a[N], b[N], c[N];
	int* d_a, * d_b, * d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	random_ints(a);
	random_ints(b);

	//a = (int *)malloc(size); 
	//b = (int *)malloc(size); random_ints(b);
	//c = (int *)malloc(size);
	//c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	// Launch add() kernel on GPU
	vectorAdd << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_a, d_b, d_c, N);
	checkCUDAError("CUDA kernel");

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Kernel Execution Time: %f ms\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	// Cleanup
	//free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	// Calculate Theoretical Bandwidth
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0); // Assuming you want to query the first CUDA device

	float theoreticalBW = (float)(prop.memoryClockRate) * 2 * prop.memoryBusWidth / (8 * 1024 * 1024 * 1024); // Convert to GB/s
	printf("Theoretical Bandwidth: %f GB/s\n", theoreticalBW);

	// Calculate Measured Bandwidth
	float RBytes = N * sizeof(int); // Number of bytes read by the kernel
	float WBytes = N * sizeof(int); // Number of bytes written by the kernel

	float measuredBW = (RBytes + WBytes) / (elapsedTime / 1000); // Convert elapsedTime to seconds
	printf("Measured Bandwidth: %f GB/s\n", measuredBW);

	return 0;
}

void checkCUDAError(const char* msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int* a)
{
	for (unsigned int i = 0; i < N; i++) {
		a[i] = rand();
	}
}
